#include <iostream>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "NeuralNetwork.cuh"
#include "LinearLayer.cuh"
#include "ReluActivation.cuh"
#include "SigmoidActivation.cuh"
#include "NNException.cuh"
#include "BCECost.cuh"

#include "CoordinatesDataset.cuh"

float computeAccuracy(const Matrix& predictions, const Matrix& targets);

int main()
{
	srand(time(NULL));

	CoordinatesDataset dataset(100, 21);
	BCECost bce_cost;

	NeuralNetwork nn;
	nn.addLayer(new LinearLayer("linear_1", Shape(2, 30)));
	nn.addLayer(new ReLUActivation("relu_1"));
	nn.addLayer(new LinearLayer("linear_2", Shape(30, 1)));
	nn.addLayer(new SigmoidActivation("sigmoid_output"));

	// Network Training
	Matrix Y;
	for (int epoch = 0; epoch < 1001; epoch++)
	{
		float cost = 0.0;

		for (int batch = 0; batch < dataset.getNumOfBatches() - 1; batch++)
		{
			Y = nn.forward(dataset.getBatches().at(batch));
			nn.backprop(Y, dataset.getTargets().at(batch));
			cost += bce_cost.cost(Y, dataset.getTargets().at(batch));
		}

		if (epoch % 1 == 0)
		{
			cout << "Epoch: " << epoch << ", Cost: " << cost / dataset.getNumOfBatches() << endl;

			/*Y = nn.forward(dataset.getBatches().at(dataset.getNumOfBatches() - 1));
			Y.copyDeviceToHost();
			cout << "--------Predictions--------" << endl;
			for (int index = 0; index < Y.shape.x; index++)
			{
				int printVal = Y[index] > 0.5 ? 1 : 0;
				cout << printVal << "\t";
			}
			cout << endl;

			cout << "--------Targets--------" << endl;
			for (int index = 0; index < dataset.getTargets().at(dataset.getNumOfBatches() - 1).shape.x; index++)
			{
				cout << dataset.getTargets().at(dataset.getNumOfBatches() - 1)[index] << "\t";
			}
			cout << endl;*/
		}
	}

	// Compute Accuracy
	Y = nn.forward(dataset.getBatches().at(dataset.getNumOfBatches() - 1));
	Y.copyDeviceToHost();

	float accuracy = computeAccuracy(Y, dataset.getTargets().at(dataset.getNumOfBatches() - 1));
	cout << "Accuracy: " << accuracy << endl;

	return 0;
}

float computeAccuracy(const Matrix& predictions, const Matrix& targets)
{
	int m = predictions.shape.x;
	int correct_predictions = 0;

	for (int i = 0; i < m; i++)
	{
		float prediction = predictions[i] > 0.5 ? 1 : 0;
		if (prediction == targets[i])
		{
			correct_predictions++;
		}
	}

	return static_cast<float>(correct_predictions) / m;
}
