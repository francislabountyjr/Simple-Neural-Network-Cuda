#include "NNException.cuh"

NNException::NNException(const char* exception_message)
	:exception_message(exception_message)
{}

const char* NNException::what() const throw()
{
	return exception_message;
}

void NNException::throwIfDeviceErrorsOccurred(const char* exception_message)
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		cerr << error << ": " << exception_message;
		throw NNException(exception_message);
	}
}
